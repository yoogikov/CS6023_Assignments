/**
*   CS6023: GPU Programming 
*   Assignment 1
*   
*   Please don't change any existing code in this file.
*
*   You can add your code whereever needed. Please add necessary memory APIs
*   for your implementation. Use cudaFree() to free up memory as soon as you're
*   done with an allocation. This will ensure that you don't run out of memory 
*   while running large test cases. Use the minimum required memory for your 
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>

using std::cin;
using std::cout;


__global__
void CalculateHadamardProduct(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
    unsigned id = blockDim.x*blockIdx.x + threadIdx.x;
    if(id >= N*N)return;
    unsigned x = id/N;
    unsigned y = id%N;
    A[x*N+y] = A[x*N+y]*B[x+N*y];
}

__global__
void FindWeightMatrix(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
    unsigned id = threadIdx.x + blockDim.x*(threadIdx.y + blockDim.y*blockIdx.x);
    if(id >= N*N)return;
    A[id] = A[id]>B[id]?A[id]:B[id];
}

__global__
void CalculateFinalMatrix(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
    unsigned id = threadIdx.x + blockDim.x*(threadIdx.y + blockDim.y*(blockIdx.x + gridDim.x*blockIdx.y));
    if(id >= 4*N*N)return;
    unsigned x = id/(2*N);
    unsigned y = id%(2*N);
    B[x*2*N+y] *= A[(x%N)*N + y%N];
}


int main(int argc, char** argv) {


    int N;
    cin >> N;
    long int* A = new long int[N * N];
    long int* B = new long int[N * N];
    long int* C = new long int[N * N];
    long int* D = new long int[2 * N * 2 * N];


    for (long int i = 0; i < N * N; i++) {
        cin >> A[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> B[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> C[i];
    }

    for (long int i = 0; i < 2 * N * 2 * N; i++) {
        cin >> D[i];
    }

    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    */

    long int* d_A;
    long int* d_B;
    long int* d_C;
    long int* d_D;

    dim3 threadsPerBlock(1024, 1, 1);
    dim3 blocksPerGrid(ceil(N * N / 1024.0), 1, 1);

    hipMalloc(&d_A, N*N*(sizeof(long int)));
    hipMalloc(&d_B, N*N*(sizeof(long int)));
    hipMalloc(&d_C, N*N*(sizeof(long int)));
    hipMalloc(&d_D, 4*N*N*(sizeof(long int)));

    hipMemcpy(d_A, A, N*N*(sizeof(long int)), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*N*(sizeof(long int)), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, N*N*(sizeof(long int)), hipMemcpyHostToDevice);
    hipMemcpy(d_D, D, 4*N*N*(sizeof(long int)), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();
    CalculateHadamardProduct<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed1 = end - start;
    
    hipFree(&d_B);

    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(N * N / 1024.0), 1, 1);


    start = std::chrono::high_resolution_clock::now();
    FindWeightMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, N);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed2 = end - start;

    hipFree(&d_C);

    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(2 * N / 32.0), ceil(2 * N / 32.0), 1);


    start = std::chrono::high_resolution_clock::now();
    CalculateFinalMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_D, N);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed3 = end - start;

    hipFree(&d_A);

    // Make sure your final output from the device is stored in d_D.

    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */

    hipMemcpy(D, d_D, 2 * N * 2 * N * sizeof(long int), hipMemcpyDeviceToHost);

    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < 2 * N; i++) {
            for (long int j = 0; j < 2 * N; j++) {
                file << D[i * 2 * N + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2 << elapsed2.count() << "\n";
        file2 << elapsed3.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}
